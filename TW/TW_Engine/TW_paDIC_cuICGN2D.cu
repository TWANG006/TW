#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime_api.h"

#include "TW_paDIC_cuICGN2D.h"

namespace TW{
namespace paDIC{

cuICGN2D::cuICGN2D(//const cv::Mat& refImg,
				   int_t iImgWidth, int_t iImgHeight,
				   int_t iStartX, int_t iStartY,
				   int_t iROIWidth, int_t iROIHeight,
				   int_t iSubsetX, int_t iSubsetY,
				   int_t iNumberX, int_t iNumberY,
				   int_t iNumIterations,
				   real_t fDeltaP,
				   ICGN2DInterpolationFLag Iflag)
			 : ICGN2D(//refImg,
					 iImgWidth, iImgHeight,
					 iStartX, iStartY,
					 iROIWidth, iROIHeight,
					 iSubsetX, iSubsetY,
					 iNumberX, iNumberY,
					 iNumIterations,
					 fDeltaP)
			  , m_Iflag(Iflag)
{
	// Allocate All needed memory
	cuICGN2D_prepare();
}


cuICGN2D::~cuICGN2D()
{}

void cuICGN2D::cuICGN2D_prepare()
{
	switch (m_Iflag)
	{
	case TW::paDIC::ICGN2DInterpolationFLag::Bicubic:
	{
		hipMalloc((void**)&g_cuHandleICGN.m_d_fRx, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_fRy, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_fTx, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_fTy, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_fTxy, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_f4InterpolationLUT, sizeof(real_t)*m_iROIWidth*m_iROIHeight * 4);

		break;
	}

	case TW::paDIC::ICGN2DInterpolationFLag::BicubicSpline:
	{
		hipMalloc((void**)&g_cuHandleICGN.m_d_fRx, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_fRy, sizeof(real_t)*m_iROIWidth*m_iROIHeight);
		hipMalloc((void**)&g_cuHandleICGN.m_d_f4InterpolationLUT, sizeof(real_t)*m_iROIWidth*m_iROIHeight * 4);
		
		break;
	}

	default:
		break;
	}

	hipMalloc((void**)&g_cuHandleICGN.m_d_fSubsetR, sizeof(real_t)*m_iPOINumber*(m_iSubsetSize));
	hipMalloc((void**)&g_cuHandleICGN.m_d_fSubsetT, sizeof(real_t)*m_iPOINumber*(m_iSubsetSize));
	hipMalloc((void**)&g_cuHandleICGN.m_d_fSubsetAveR, sizeof(real_t)*m_iPOINumber*(m_iSubsetSize + 1));
	hipMalloc((void**)&g_cuHandleICGN.m_d_fSubsetAveT, sizeof(real_t)*m_iPOINumber*(m_iSubsetSize + 1));
	hipMalloc((void**)&g_cuHandleICGN.m_d_Hessian, sizeof(real_t)*m_iPOINumber * 6 * 6);
	hipMalloc((void**)&g_cuHandleICGN.m_d_Hessian, sizeof(real_t)*m_iPOINumber * m_iSubsetSize * 6);
	hipMalloc((void**)&g_cuHandleICGN.m_d_iIterationNums, sizeof(int)*m_iPOINumber);
}

}// namespace paDIC
}// namespace TW
