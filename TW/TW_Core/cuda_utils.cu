#include <cuda_utils.h>

#include <hip/hip_runtime.h>
#include <>




void updatePOI_ROI(int *iPOIpos,
				   int *iU,
				   int *iV,
				   int iSubsetX,
				   int iSubsetY,
				   int iMarginX,
				   int iMarginY,
				   int &iStartX,
				   int &iStartY,
				   int &iROIWidth,
				   int &iROIHeight)
{

}